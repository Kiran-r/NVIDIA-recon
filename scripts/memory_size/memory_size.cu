#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define cudamalloc(p, size) {                                               \
    hipMalloc(&p, size);                                                   \
    if (p)                                                                  \
        printf("Allocated %zu bytes from %p \n", size, p);                  \
    else                                                                    \
        printf("Failed to allocate %zu bytes\n", size);                     \
}

int main()
{
    size_t step = 0x1000000;
    size_t size = step;
    static size_t best = 0;
    hipError_t e;
    
    while (step > 0)
    {
        void *p;

        //Try allocating Memory
        cudamalloc(p, size);
        e = hipGetLastError();

        //Check if successful
        if (e==hipSuccess) {
            hipFree(p);
            best = size;
        }
        else {
            step /= 0x10;
        }
        size += step;
    }

    void *p;
    //Confirm
    cudamalloc(p, best);
    e = hipGetLastError();
    if (e==hipSuccess)
    {
        printf("\nBest possible allocatable block size is %.4f GB\n",
               (float)best/1000000000.0);
        hipFree(p);
        return 0;
    }
    else
        return 1;
}
